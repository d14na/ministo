#include <assert.h>
#include <sstream>
#include <iomanip>
#include <stdio.h>
#include <iostream>
#include <string.h>

#include "solver.h"
// NOTE: We will need this!
#include "sha3.cu"

#include "../utilities.cpp"

// static
std::atomic<uint32_t> CUDASolver::hashes(0u); // statistics only


/**
 * CUDA Solver
 */
CUDASolver::CUDASolver() noexcept :
    m_address(ADDRESS_LENGTH),
    m_challenge(UINT256_LENGTH),
    m_target(UINT256_LENGTH),
    m_target_tmp(UINT256_LENGTH),
    m_buffer(ADDRESS_LENGTH + 2 * UINT256_LENGTH),
    m_buffer_tmp(ADDRESS_LENGTH + 2 * UINT256_LENGTH), //this has something to do with updateBuffer
    m_buffer_ready(false),
    m_target_ready(false),
    m_updated_gpu_inputs(false)
{ }

/**
 * Set Address
 */
void CUDASolver::setAddress(std::string const& addr)
{
    std::cout << "Setting CUDA Address\n";

    assert(addr.length() == (ADDRESS_LENGTH * 2 + 2));

    hexToBytes(addr, m_address);

    m_updated_gpu_inputs = true;

    updateGPULoop();
}

/**
 * Set Challenge
 */
void CUDASolver::setChallenge(std::string const& chal)
{
    std::cout << "Setting CUDA Challenge\n";

    s_challenge = chal;

    assert(chal.length() == (UINT256_LENGTH * 2 + 2));

    hexToBytes(chal, m_challenge);

    m_updated_gpu_inputs = true;

    updateGPULoop();
}

void CUDASolver::setTarget(std::string const& target)
{
    std::cout << "Setting CUDA Target [ " << target << " ]\n";

    assert(target.length() <= (UINT256_LENGTH * 2 + 2));

    std::string const t(static_cast<std::string::size_type>(UINT256_LENGTH * 2 + 2) - target.length(), '0');

    s_target = target;

    /**
     * Double-buffer system, the trySolution() function will be blocked
     * only when a change occurs.
     */
    {
        std::lock_guard<std::mutex> g(m_target_mutex);
        hexToBytes("0x" + t + target.substr(2), m_target_tmp);
    }

    /* Set target flag. */
    m_target_ready = true;

    m_updated_gpu_inputs = true;

    updateGPULoop();
}

/**
 * Set Block Size
 */
void CUDASolver::setBlockSize(int size)
{
    std::cout << "CUDASolver: Setting BlockSize [ " << size << " ]\n";

    setCudaBlockSize(size);
}

/**
 * Set Thread Size
 */
void CUDASolver::setThreadSize(int size)
{
    std::cout << "CUDASolver: Setting Thread Size [ " << size << " ]\n";

    setCudaThreadSize(size);
}

/**
 * Requires Restart
 */
bool CUDASolver::requiresRestart()
{
    return m_updated_gpu_inputs;
}

/**
 * Update GPU Loop
 *
 * NOTE: This will restart the miner if needed.
 */
void CUDASolver::updateGPULoop()
{
    if (m_updated_gpu_inputs
        && m_target_ready
        && m_challenge.size() > 0
        && m_address.size() > 0 ) {
        /* Set update flag. */
        m_updated_gpu_inputs = false;

        printf("Target input:\n");

        if (s_target.length() < 66) {
            std::string zeros = std::string(66 - s_target.length(), '0');

            std::string s = "0x" + zeros + s_target.substr(2, s_target.length());

            s_target = s;
        }

        unsigned char target_input[64];

        bytes_t target_bytes(32);

        hexToBytes(s_target, target_bytes);

        for (int i = 0; i < 32; i++) {
            target_input[i] =(unsigned char) target_bytes[i];

            printf("%02x",(unsigned char) target_input[i]);
        }

        unsigned char hash_prefix[52];

        std::string clean_challenge = s_challenge;

        bytes_t challenge_bytes(32);

        hexToBytes(clean_challenge, challenge_bytes);

        for (int i = 0; i < 32; i++) {
            hash_prefix[i] = (unsigned char) challenge_bytes[i];
        }

        for (int i = 0; i < 20; i++) {
            hash_prefix[i+32] = (unsigned char)m_address[i];
        }

        printf("Challenge+Address:\n");

        for (int i = 0; i < 52; i++) {
            printf("%02x", (unsigned char) hash_prefix[i]);
        }

        printf("\n/prefix\n");

        printf("Updating mining inputs..\n");

        update_mining_inputs((const char *)target_input , (const char *)hash_prefix);
    }
}

/**
 * Init
 *
 * NOTE: Call the sha3.cu init func.
 */
void CUDASolver::init()
{
    std::cout << "CUDASolver Initializing.. \n ";

    gpu_init();
}

/**
 * Stop Solving
 */
void CUDASolver::stopSolving( )
{
    std::cout << "CUDA has stopped solving for now.\n ";

    /* Stop solving. */
    stop_solving();
}

/**
 * Find Solution
 */
CUDASolver::bytes_t CUDASolver::findSolution( )
{
    /* Set update flag. */
    m_updated_gpu_inputs = false;

    std::cout << "CUDA is trying to find a solution :) \n ";

    hipEventCreate(&start);

    hipEventCreate(&stop);

    printf("Target input:\n");

    if (s_target.length() < 66) {
        std::string zeros = std::string(66 - s_target.length(), '0');

        std::string s = "0x" + zeros + s_target.substr(2, s_target.length());

        s_target = s;
    }

    unsigned char target_input[64];

    bytes_t target_bytes(32);

    hexToBytes(s_target, target_bytes);

    for (int i = 0; i < 32; i++) {
  	     target_input[i] = (unsigned char)target_bytes[i];

         printf("%02x", (unsigned char)target_input[i]);
     }

     unsigned char hash_prefix[52];

     std::string clean_challenge = s_challenge;

     bytes_t challenge_bytes(32);

     hexToBytes(clean_challenge, challenge_bytes);

     for (int i = 0; i < 32; i++) {
         hash_prefix[i] = (unsigned char)challenge_bytes[i];
     }

     for (int i = 0; i < 20; i++) {
         hash_prefix[i+32] = (unsigned char)m_address[i];
     }

     printf("Challenge+Address:\n");

     for (int i = 0; i < 52; i++) {
         printf("%02x", (unsigned char)hash_prefix[i]);
     }

     printf("\n/prefix\n");

     unsigned char * s_solution = find_message((const char *)target_input, (const char *)hash_prefix);

     //here

     CUDASolver::bytes_t byte_solution(32);

     for (int i = 52; i < 84; i++) {
         byte_solution[i - 52] = (uint8_t)s_solution[i];

         //std::cout << (uint8_t)s_solution[i] << "\n";
     }

     hipEventDestroy(start);

     hipEventDestroy(stop);

     return byte_solution;
}

/**
 * Hex String
 */
std::string CUDASolver::hexStr( char* data, int len)
{
    std::stringstream ss;

    ss << std::hex;

    for (int i = 0; i < len; ++i)
        ss << std::setw(2) << std::setfill('0') << (int)data[i];

    return ss.str();
}

/**
 * Hex To Bytes
 *
 * NOTE: This is a static function.
 */
void CUDASolver::hexToBytes(std::string const& hex, bytes_t& bytes)
{
    assert(hex.length() % 2 == 0);
    assert(bytes.size() == (hex.length() / 2 - 1));

    HexToBytes(hex.substr(2), &bytes[0]);
}

/**
 * Bytes to String
 *
 * NOTE: This is a static function.
 */
std::string CUDASolver::bytesToString(bytes_t const& buffer)
{
    std::string output;

    output.reserve(buffer.size() * 2 + 1);

    for (unsigned i = 0; i < buffer.size(); ++i)
        output += ascii[buffer[i]];

    return output;
}

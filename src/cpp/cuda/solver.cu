#include <assert.h>
#include <sstream>
#include <iomanip>
#include <stdio.h>
#include <iostream>
#include <string.h>

#include "solver.h"
// NOTE: We will need this!
// #include "sha3.h"
#include "sha3.cu"

#include "../utilities.cpp"

// static
std::atomic<uint32_t> CUDASolver::hashes(0u); // statistics only


/**
 * CUDA Solver
 */
CUDASolver::CUDASolver() noexcept :
    m_address(ADDRESS_LENGTH),
    m_challenge(UINT256_LENGTH),
    m_target(UINT256_LENGTH),
    m_target_tmp(UINT256_LENGTH),
    m_buffer(ADDRESS_LENGTH + 2 * UINT256_LENGTH),
    m_buffer_tmp(ADDRESS_LENGTH + 2 * UINT256_LENGTH), //this has something to do with updateBuffer
    m_buffer_ready(false),
    m_target_ready(false),
    m_updated_gpu_inputs(false)
{ }

/**
 * Set Address
 */
void CUDASolver::setAddress(std::string const& addr)
{
    std::cout << "Setting CUDA Address\n";

    /* Validate address length. */
    assert(addr.length() == (ADDRESS_LENGTH * 2 + 2));

    /* Convert from hex to bytes. */
    hexToBytes(addr, m_address);

    /* Set GPU inputs flag. */
    m_updated_gpu_inputs = true;

    /* Update GPU loop. */
    updateGPULoop();
}

/**
 * Set Challenge
 */
void CUDASolver::setChallenge(std::string const& chal)
{
    std::cout << "Setting CUDA Challenge\n";

    /* Validate challenge length. */
    assert(chal.length() == (UINT256_LENGTH * 2 + 2));

    /* Set challenge string. */
    s_challenge = chal;

    /* Convert from hex to bytes. */
    hexToBytes(chal, m_challenge);

    /* Set GPU inputs flag. */
    m_updated_gpu_inputs = true;

    /* Update GPU loop. */
    updateGPULoop();
}

void CUDASolver::setTarget(std::string const& target)
{
    std::cout << "Setting CUDA Target [ " << target << " ]\n";

    assert(target.length() <= (UINT256_LENGTH * 2 + 2));

    std::string const t(static_cast<std::string::size_type>(UINT256_LENGTH * 2 + 2) - target.length(), '0');

    s_target = target;

    /**
     * Double-buffer system, the trySolution() function will be blocked
     * only when a change occurs.
     */
    {
        std::lock_guard<std::mutex> g(m_target_mutex);

        /* Convert from hex to bytes. */
        hexToBytes("0x" + t + target.substr(2), m_target_tmp);
    }

    /* Set target flag. */
    m_target_ready = true;

    /* Set GPU inputs flag. */
    m_updated_gpu_inputs = true;

    /* Update GPU loop. */
    updateGPULoop();
}

/**
 * Set Block Size
 */
void CUDASolver::setBlockSize(int size)
{
    std::cout << "CUDASolver: Setting BlockSize [ " << size << " ]\n";

    setCudaBlockSize(size);
}

/**
 * Set Thread Size
 */
void CUDASolver::setThreadSize(int size)
{
    std::cout << "CUDASolver: Setting Thread Size [ " << size << " ]\n";

    setCudaThreadSize(size);
}

/**
 * Requires Restart
 */
bool CUDASolver::requiresRestart()
{
    return m_updated_gpu_inputs;
}

/**
 * Update GPU Loop
 *
 * NOTE: This will restart the miner if needed.
 */
void CUDASolver::updateGPULoop()
{
    if (m_updated_gpu_inputs
        && m_target_ready
        && m_challenge.size() > 0
        && m_address.size() > 0) {
        /* Set GPU inputs flag. */
        m_updated_gpu_inputs = false;

        printf("Target input:\n");

        /* Validate target length. */
        if (s_target.length() < 66) {
            /* Calculate zero padding (if necessary). */
            std::string zeros = std::string(66 - s_target.length(), '0');

            /* Add zero padding (if necessary). */
            std::string s = "0x" + zeros + s_target.substr(2, s_target.length());

            /* Re-assign target (string). */
            s_target = s;
        }

        /* Initialize target input. */
        unsigned char target_input[64];

        /* Initialize target bytes. */
        bytes_t target_bytes(32);

        /* Convert from hex to bytes. */
        hexToBytes(s_target, target_bytes);

        for (int i = 0; i < 32; i++) {
            target_input[i] = (unsigned char) target_bytes[i];

            printf("%02x",(unsigned char) target_input[i]);
        }

        unsigned char hash_prefix[52];

        std::string clean_challenge = s_challenge;

        bytes_t challenge_bytes(32);

        /* Convert from hex to bytes. */
        hexToBytes(clean_challenge, challenge_bytes);

        for (int i = 0; i < 32; i++) {
            hash_prefix[i] = (unsigned char) challenge_bytes[i];
        }

        for (int i = 0; i < 20; i++) {
            hash_prefix[i+32] = (unsigned char)m_address[i];
        }

        printf("Challenge+Address:\n");

        for (int i = 0; i < 52; i++) {
            printf("%02x", (unsigned char) hash_prefix[i]);
        }

        printf("\n/prefix\n");

        printf("Updating mining inputs..\n");

        update_mining_inputs((const char *)target_input , (const char *)hash_prefix);
    }
}

/**
 * Init
 *
 * NOTE: Call the sha3.cu init func.
 */
void CUDASolver::init()
{
    std::cout << "CUDASolver Initializing.. \n ";

    /* GPU initialization. */
    gpu_init();
}

/**
 * Stop Solving
 */
void CUDASolver::stopSolving( )
{
    std::cout << "CUDA has stopped solving for now.\n ";

    /* Stop solving. */
    stop_solving();
}

/**
 * Find Solution
 */
CUDASolver::bytes_t CUDASolver::findSolution( )
{
    std::cout << "CUDA is trying to find a solution :)\n ";

    /* Set GPU inputs flag. */
    m_updated_gpu_inputs = false;

    /* Create CUDA events. */
    hipEventCreate(&start);
    hipEventCreate(&stop);

    printf("Target input:\n");

    /* Validate target length. */
    if (s_target.length() < 66) {
        /* Calculate zero padding (if necessary). */
        std::string zeros = std::string(66 - s_target.length(), '0');

        /* Add zero padding (if necessary). */
        std::string s = "0x" + zeros + s_target.substr(2, s_target.length());

        /* Re-assign target (string). */
        s_target = s;
    }

    /* Initialize target input. */
    unsigned char target_input[64];

    /* Initialize target bytes. */
    bytes_t target_bytes(32);

    /* Convert from hex to bytes. */
    hexToBytes(s_target, target_bytes);

    /* Copy target `target_bytes` to `target_input`. */
    for (int i = 0; i < 32; i++) {
  	     target_input[i] = (unsigned char)target_bytes[i];

         // printf("%02x", (unsigned char)target_input[i]);
     }

     /* Initialize hash prefix. */
     unsigned char hash_prefix[52];

     /* Copy challenge (string) to `clean_challenge`. */
     std::string clean_challenge = s_challenge;

     /* Initialize challenge bytes. */
     bytes_t challenge_bytes(32);

     /* Convert from hex to bytes. */
     hexToBytes(clean_challenge, challenge_bytes);

     /* Copy challenge (bytes) to `hash_prefix`. */
     for (int i = 0; i < 32; i++) {
         hash_prefix[i] = (unsigned char)challenge_bytes[i];
     }

     /* Copy address (bytes) to `hash_prefix`. */
     // NOTE: Placed after the challenge.
     for (int i = 0; i < 20; i++) {
         hash_prefix[i + 32] = (unsigned char)m_address[i];
     }

     printf("Challenge + Address:\n");

     for (int i = 0; i < 52; i++) {
         printf("%02x", (unsigned char)hash_prefix[i]);
     }

     printf("\n/Prefix\n");

     unsigned char * s_solution = find_message(
         (const char *)target_input, (const char *)hash_prefix);

     // here

     /* Initialize solution (vector). */
     CUDASolver::bytes_t byte_solution(32);

     /* Copy solution to vector. */
     for (int i = 52; i < 84; i++) {
         byte_solution[i - 52] = (uint8_t)s_solution[i];

         // std::cout << (uint8_t)s_solution[i] << "\n";
     }

     /* Destroy CUDA events. */
     hipEventDestroy(start);
     hipEventDestroy(stop);

     /* Return solution. */
     return byte_solution;
}

/**
 * Hex String
 */
std::string CUDASolver::hexStr( char* data, int len)
{
    std::stringstream ss;

    ss << std::hex;

    for (int i = 0; i < len; ++i)
        ss << std::setw(2) << std::setfill('0') << (int)data[i];

    return ss.str();
}

/**
 * Hex To Bytes
 *
 * NOTE: This is a static function.
 */
void CUDASolver::hexToBytes(std::string const& hex, bytes_t& bytes)
{
    assert(hex.length() % 2 == 0);
    assert(bytes.size() == (hex.length() / 2 - 1));

    /* Call Utilities::HexToBytes. */
    HexToBytes(hex.substr(2), &bytes[0]);
}

/**
 * Bytes to String
 *
 * NOTE: This is a static function.
 */
std::string CUDASolver::bytesToString(bytes_t const& buffer)
{
    std::string output;

    output.reserve(buffer.size() * 2 + 1);

    for (unsigned i = 0; i < buffer.size(); ++i)
        output += ascii[buffer[i]];

    return output;
}
